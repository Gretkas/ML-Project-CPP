
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void ojas_rule(float *x, float *w, const float y, const float learning_rate) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  float temp = x[i] - y * w[i];
  w[i] = w[i] + learning_rate * y * temp;
};

int main() {
  const int n = 0; //length of arrays
  const float y = 0;
  const float learning_rate = 0;
  float *w, *x;
  float *c_w, *c_x;

  //Initilaisere verdier mellom 0 og 1

  hipMalloc(&c_w, sizeof(w));
  hipMalloc(&c_x, sizeof(x));

  hipMemcpy(c_w, w, sizeof(w), hipMemcpyHostToDevice);
  hipMemcpy(c_x, x, sizeof(x), hipMemcpyHostToDevice);

  int block_size = 256;                          //number of threads per block
  int grid_size = (n + block_size) / block_size; //number of blocks
  ojas_rule<<<grid_size, block_size>>>(c_x, c_w, y, learning_rate);

  hipDeviceSynchronize();
  hipMemcpy(w, c_w, sizeof(w), hipMemcpyDeviceToHost);

  hipFree(c_w);
  hipFree(c_x);
  free(w);
  free(x);
}
