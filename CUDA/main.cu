#include "hip/hip_runtime.h"
#include "../src/MNIST/mnist_loader.cpp"
#include <iostream>
#include <random>
#include <typeinfo>

using namespace std;

__global__ void ojas_rule(float *x, float *w, const float y, const float learning_rate) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = x[i] - y * w[i];
    w[i] = w[i] + learning_rate * y * temp;
};

//Burde denne kjøres på divice eller på
__device__ float y(const float *x, const float *w, int length) {
    float y = 0;
    for (int i = 0; i < length; ++i) {
        y += w[i] * x[i];
    }
    return y;
}

__host__ void run_ojas(float *w, const float *x) {

    const int n = 0; //length of arrays
    const float y = 0;
    const float learning_rate = 0;
    float *c_w, *c_x;

    //Initilaisere verdier mellom 0 og 1

    hipMalloc(&c_w, sizeof(w)); // er sizeof her riktig?? Ja er vel det
    hipMalloc(&c_x, sizeof(x));

    hipMemcpy(c_w, w, sizeof(w), hipMemcpyHostToDevice);
    hipMemcpy(c_x, x, sizeof(x), hipMemcpyHostToDevice);

    int block_size = 256;                          //number of threads per block
    int grid_size = (n + block_size) / block_size; //number of blocks
    ojas_rule<<<grid_size, block_size>>>(c_x, c_w, y, learning_rate);

    hipDeviceSynchronize();
    hipMemcpy(w, c_w, sizeof(w), hipMemcpyDeviceToHost);

    hipFree(c_w);
    hipFree(c_x);
}

__host__ float *load_data() {
    mnist_loader train("datasets/train-images.idx3-ubyte",
                       "datasets/train-labels.idx1-ubyte", 100);
    mnist_loader test("datasets/t10k-images.idx3-ubyte",
                      "datasets/t10k-labels.idx1-ubyte", 100);

    std::vector<float> img_seg = train.image_segment();

    float *x;
    x = img_seg.data();
    return x;
}

//Husk å free arrayet etter bruk!!
__host__ float *generate_w(const int len) {
    float *w;
    w = new float[len];
    srand((unsigned)time(0));

    for (int i = 0; i < len; ++i) {
        w[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX / (1 + 1))) - 1.0;
    }
    return w;
}

//må free w og x;
int main() {

    int len = 25; //lengden på diverse arrays
    float *w = generate_w(len);

    float *x = load_data();

    for (int i = 0; i < len; ++i) {
        cout << x[i] << endl;
    }
    //run_ojas(w, x);

    //Får følgende feil men å ha disse:
    //free(): double free detected in tcache 2
    //delete x;
    //delete w;
}
